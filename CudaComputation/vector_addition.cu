#include "hip/hip_runtime.h"
﻿#include <exception>
#include <string>

#include "cuda_computation.hpp"
#include "hip/hip_runtime.h"
#include ""

__global__ void single_precision_vector_addition_kernel(
	const float* left_vector,
	const float* right_vector,
	float* result_vector)
{
	const unsigned int element_index = threadIdx.x;

	result_vector[element_index] = left_vector[element_index] + right_vector[element_index];
}

extern "C" __declspec(dllexport) computation_result single_precision_vector_addition(
	const float* left_vector,
	const float* right_vector,
	float* result_vector,
	const unsigned long vector_length)
{
	try
	{
		throw_on_cuda_error(hipSetDevice(0), cuda_set_device_failed);
		throw_on_cuda_error(hipDeviceReset(), cuda_device_reset_failed);

		const size_t buffer_size = vector_length * sizeof(float);

		const float_vector_in_device_memory left_vector_in_device_memory(vector_length);
		const float_vector_in_device_memory right_vector_in_device_memory(vector_length);
		const float_vector_in_device_memory result_vector_in_device_memory(vector_length);		

		throw_on_cuda_error(hipMemcpy(left_vector_in_device_memory.device_pointer, left_vector, buffer_size, hipMemcpyHostToDevice), cuda_memcpy_failed);
		throw_on_cuda_error(hipMemcpy(right_vector_in_device_memory.device_pointer, right_vector, buffer_size, hipMemcpyHostToDevice), cuda_memcpy_failed);

		single_precision_vector_addition_kernel<<<1, vector_length>>> (
			left_vector_in_device_memory.device_pointer, 
			right_vector_in_device_memory.device_pointer,
			result_vector_in_device_memory.device_pointer
			);

		throw_on_cuda_error(hipGetLastError(), cuda_kernel_failed);
		throw_on_cuda_error(hipDeviceSynchronize(), cuda_device_synchronize_failed);

		throw_on_cuda_error(hipMemcpy(result_vector, result_vector_in_device_memory.device_pointer, buffer_size, hipMemcpyDeviceToHost), cuda_memcpy_failed);

		return succeeded;
	}
	catch (const computation_failed_exception& exception)
	{
		return exception.failure;
	}
}

__global__ void double_precision_vector_addition_kernel(
	const double* left_vector,
	const double* right_vector,
	double* result_vector)
{
	const unsigned int element_index = threadIdx.x;

	result_vector[element_index] = left_vector[element_index] + right_vector[element_index];
}

extern "C" __declspec(dllexport) computation_result double_precision_vector_addition(
	const double* left_vector,
	const double* right_vector,
	double* result_vector,
	const unsigned long vector_length)
{
	try
	{
		throw_on_cuda_error(hipSetDevice(0), cuda_set_device_failed);
		throw_on_cuda_error(hipDeviceReset(), cuda_device_reset_failed);

		const double_vector_in_device_memory left_vector_in_device_memory(vector_length);
		const double_vector_in_device_memory right_vector_in_device_memory(vector_length);
		const double_vector_in_device_memory result_vector_in_device_memory(vector_length);

		const size_t buffer_size = vector_length * sizeof(double);

		throw_on_cuda_error(hipMemcpy(left_vector_in_device_memory.device_pointer, left_vector, buffer_size, hipMemcpyHostToDevice), cuda_memcpy_failed);
		throw_on_cuda_error(hipMemcpy(right_vector_in_device_memory.device_pointer, right_vector, buffer_size, hipMemcpyHostToDevice), cuda_memcpy_failed);

		double_precision_vector_addition_kernel<<<1, vector_length>>> (
			left_vector_in_device_memory.device_pointer,
			right_vector_in_device_memory.device_pointer,
			result_vector_in_device_memory.device_pointer
			);

		throw_on_cuda_error(hipGetLastError(), cuda_kernel_failed);
		throw_on_cuda_error(hipDeviceSynchronize(), cuda_device_synchronize_failed);

		throw_on_cuda_error(hipMemcpy(result_vector, result_vector_in_device_memory.device_pointer, buffer_size, hipMemcpyDeviceToHost), cuda_memcpy_failed);

		return succeeded;
	}
	catch (const computation_failed_exception& exception)
	{
		return exception.failure;
	}
}
