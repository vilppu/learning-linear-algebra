#include "hip/hip_runtime.h"
﻿#include <exception>
#include <iostream>

#include "cuda_computation.hpp"
#include "hip/hip_runtime.h"
#include ""

template<typename TRealNumber>
__global__ void vector_addition_kernel(
	const TRealNumber left_vector[],
	const TRealNumber right_vector[],
	TRealNumber* result_vector,
	const unsigned long vector_length)
{
	const auto i = blockIdx.x * blockDim.x;
	const auto element_index = i + threadIdx.x;

	if (element_index < vector_length) {
		result_vector[element_index] = left_vector[element_index] + right_vector[element_index];
	}
}

template<typename TRealNumber>
int vector_addition(
	const TRealNumber left_vector[],
	const TRealNumber right_vector[],
	TRealNumber* result_vector,
	const unsigned long vector_length)
{
	try
	{
		throw_on_cuda_error(hipSetDevice(0), cuda_set_device_failed);
		throw_on_cuda_error(hipDeviceReset(), cuda_device_reset_failed);

		const auto buffer_size = vector_length * sizeof(TRealNumber);

		const vector_in_device_memory<TRealNumber> left_vector_in_device_memory(vector_length);
		const vector_in_device_memory<TRealNumber> right_vector_in_device_memory(vector_length);
		const vector_in_device_memory<TRealNumber> result_vector_in_device_memory(vector_length);

		throw_on_cuda_error(hipMemcpy(left_vector_in_device_memory.device_pointer, left_vector, buffer_size, hipMemcpyHostToDevice), cuda_memcpy_failed);
		throw_on_cuda_error(hipMemcpy(right_vector_in_device_memory.device_pointer, right_vector, buffer_size, hipMemcpyHostToDevice), cuda_memcpy_failed);

		auto threads_per_block = 1024;
		auto blocks_per_grid = (vector_length + threads_per_block - 1) / threads_per_block;

		vector_addition_kernel<TRealNumber> << <blocks_per_grid, threads_per_block >> > (
			left_vector_in_device_memory.device_pointer,
			right_vector_in_device_memory.device_pointer,
			result_vector_in_device_memory.device_pointer,
			vector_length
			);

		throw_on_cuda_error(hipGetLastError(), cuda_kernel_failed);
		throw_on_cuda_error(hipDeviceSynchronize(), cuda_device_synchronize_failed);

		throw_on_cuda_error(hipMemcpy(result_vector, result_vector_in_device_memory.device_pointer, buffer_size, hipMemcpyDeviceToHost), cuda_memcpy_failed);

		return succeeded;
	}
	catch (const computation_failed_exception& exception)
	{
		return exception.failure;
	}
}

extern "C" __declspec(dllexport) int single_precision_vector_addition(
	const float* left_vector,
	const float* right_vector,
	float* result_vector,
	const unsigned long vector_length)
{
	return vector_addition<float>(left_vector, right_vector, result_vector, vector_length);
}

extern "C" __declspec(dllexport) int double_precision_vector_addition(
	const double* left_vector,
	const double* right_vector,
	double* result_vector,
	const unsigned long vector_length)
{
	return vector_addition<double>(left_vector, right_vector, result_vector, vector_length);
}