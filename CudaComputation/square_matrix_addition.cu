#include "hip/hip_runtime.h"
#include <exception>
#include <iostream>

#include "cuda_computation.hpp"
#include "hip/hip_runtime.h"
#include ""

template<typename TRealNumber>
__global__ void square_matrix_addition_kernel(
	TRealNumber left_matrix[],
	TRealNumber right_matrix[],
	TRealNumber result_matrix[],
	const unsigned long dimension_of_matrix)
{
	const auto column = threadIdx.x + blockDim.x * blockIdx.x;
	const auto row = threadIdx.y + blockDim.y * blockIdx.y;

	auto element_index = row * dimension_of_matrix + column;

	if (column < dimension_of_matrix && row < dimension_of_matrix) {
		result_matrix[element_index] = left_matrix[element_index] + right_matrix[element_index];
	}
}

template<typename TRealNumber>
int square_matrix_addition(
	const TRealNumber left_matrix[],
	const TRealNumber right_matrix[],
	TRealNumber result_matrix[],
	const unsigned long dimension_of_matrix)
{
	try
	{
		throw_on_cuda_error(hipSetDevice(0), cuda_set_device_failed);
		throw_on_cuda_error(hipDeviceReset(), cuda_device_reset_failed);

		const auto number_of_elements_in_matrix = dimension_of_matrix * dimension_of_matrix;
		const auto buffer_size = number_of_elements_in_matrix * sizeof(TRealNumber);

		const matrix_in_device_memory<TRealNumber> left_matrix_in_device_memory(dimension_of_matrix);
		const matrix_in_device_memory<TRealNumber> right_matrix_in_device_memory(dimension_of_matrix);
		const matrix_in_device_memory<TRealNumber> result_matrix_in_device_memory(dimension_of_matrix);

		throw_on_cuda_error(hipMemcpy(left_matrix_in_device_memory.device_pointer, left_matrix, buffer_size, hipMemcpyHostToDevice), cuda_memcpy_failed);
		throw_on_cuda_error(hipMemcpy(right_matrix_in_device_memory.device_pointer, right_matrix, buffer_size, hipMemcpyHostToDevice), cuda_memcpy_failed);

		dim3 threads_per_block(std::min(dimension_of_matrix, 32ul), std::min(dimension_of_matrix, 32ul));
		dim3 blocks_per_grid(dimension_of_matrix / threads_per_block.x, dimension_of_matrix / threads_per_block.y);

		square_matrix_addition_kernel<TRealNumber><<<blocks_per_grid, threads_per_block>>>(
			left_matrix_in_device_memory.device_pointer,
			right_matrix_in_device_memory.device_pointer,
			result_matrix_in_device_memory.device_pointer,
			dimension_of_matrix
			);

		throw_on_cuda_error(hipGetLastError(), cuda_kernel_failed);
		throw_on_cuda_error(hipDeviceSynchronize(), cuda_device_synchronize_failed);

		throw_on_cuda_error(hipMemcpy(result_matrix, result_matrix_in_device_memory.device_pointer, buffer_size, hipMemcpyDeviceToHost), cuda_memcpy_failed);

		return succeeded;
	}
	catch (const computation_failed_exception& exception)
	{
		return exception.failure;
	}
}

extern "C" __declspec(dllexport) int single_precision_square_matrix_addition(
	const float* left_matrix,
	const float* right_matrix,
	float* result_matrix,
	const unsigned long dimension_of_matrix)
{
	return square_matrix_addition<float>(left_matrix, right_matrix, result_matrix, dimension_of_matrix);
}

extern "C" __declspec(dllexport) int double_precision_square_matrix_addition(
	const double* left_matrix,
	const double* right_matrix,
	double* result_matrix,
	const unsigned long dimension_of_matrix)
{
	return square_matrix_addition<double>(left_matrix, right_matrix, result_matrix, dimension_of_matrix);
}
